
#include <hip/hip_runtime.h>
//#include "stdafx.h"
//#include "kernel.h"
//
//
//__global__ void 
//cal_val( 
//	const float* value,
//	const float c_puct, 
//	const float* p,
//	const int* visit_count,
//	const int* parent_node_visitcount,
//	float* result)
//{
//	int i = threadIdx.x;
//	result[i] = value[i] + c_puct * p[i]* parent_node_visitcount[i] / (1 + visit_count[i] );
//}
//
//__global__ void 
//get_max(float* g_idata, int*index, unsigned int n)
//{
//	unsigned int tid = threadIdx.x;
//	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
//	// convert global data pointer to the local point of this block
//	float* idata = g_idata + blockIdx.x * blockDim.x;
//	if (idx > n)
//		return;
//	//in-place reduction in global memory
//	for (int stride = 1; stride < blockDim.x; stride *= 2)
//	{
//		//convert tid into local array index
//		int index = 2 * stride * tid;
//		if (index < blockDim.x)
//		{
//			if (idata[index] < idata[index + stride])
//				idata[index] = idata[index + stride];
//
//		}
//		__syncthreads();
//	}
//	//write result for this block to global men
//	//if (tid == 0)
//	//	g_odata[blockIdx.x] = idata[0];
//}
//
//
////int ma1in()
////{
////	const int arraySize = 5;
////	const int a[arraySize] = { 1, 2, 3, 4, 5 };
////	const int b[arraySize] = { 10, 20, 30, 40, 50 };
////	int c[arraySize] = { 0 };
////
////	// Add vectors in parallel.
////	cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "addWithCuda failed!");
////		return 1;
////	}
////
////	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
////		c[0], c[1], c[2], c[3], c[4]);
////
////	// cudaDeviceReset must be called before exiting in order for profiling and
////	// tracing tools such as Nsight and Visual Profiler to show complete traces.
////	cudaStatus = cudaDeviceReset();
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaDeviceReset failed!");
////		return 1;
////	}
////
////	return 0;
////}
//
//// Helper function for using CUDA to add vectors in parallel.
////cudaError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
////{
////	int* dev_a = 0;
////	int* dev_b = 0;
////	int* dev_c = 0;
////	cudaError_t cudaStatus;
////
////	// Choose which GPU to run on, change this on a multi-GPU system.
////	cudaStatus = cudaSetDevice(0);
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
////		goto Error;
////	}
////
////	// Allocate GPU buffers for three vectors (two input, one output)    .
////	cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMalloc failed!");
////		goto Error;
////	}
////
////	cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMalloc failed!");
////		goto Error;
////	}
////
////	cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMalloc failed!");
////		goto Error;
////	}
////
////	// Copy input vectors from host memory to GPU buffers.
////	cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMemcpy failed!");
////		goto Error;
////	}
////
////	cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMemcpy failed!");
////		goto Error;
////	}
////
////	// Launch a kernel on the GPU with one thread for each element.
////	addKernel << <1, size >> > (dev_c, dev_a, dev_b);
////
////	// Check for any errors launching the kernel
////	cudaStatus = cudaGetLastError();
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
////		goto Error;
////	}
////
////	// cudaDeviceSynchronize waits for the kernel to finish, and returns
////	// any errors encountered during the launch.
////	cudaStatus = cudaDeviceSynchronize();
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
////		goto Error;
////	}
////
////	// Copy output vector from GPU buffer to host memory.
////	cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
////	if (cudaStatus != cudaSuccess) {
////		fprintf(stderr, "cudaMemcpy failed!");
////		goto Error;
////	}
////
////Error:
////	cudaFree(dev_c);
////	cudaFree(dev_a);
////	cudaFree(dev_b);
////
////	return cudaStatus;
////}
////
