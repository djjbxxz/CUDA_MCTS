#include "kernel.h"

int CUDA_accelerator::get_max_UBC_index()
{
	collect_data();
	compute();
	return max_index;
}
//for DEBUG
void check_container_device(thrust::device_vector<float> container,int print_num=3)
{
	thrust::host_vector<float> t(print_num);
	cout << "calculated device result:";
	for (int i = 0; i < print_num; i++)
	{
		t[i] = container[i];
		cout << t[i] << "    ";
	}
	cout << endl;
}
//for DEBUG
void check_container_host(thrust::host_vector<float> container, int print_num = 3)
{
	cout << "calculated host result:";
	for (int i = 0; i < print_num; i++)
	{
		cout << container[i] << "    ";
	}
	cout << endl;
}

void CUDA_accelerator::compute()
{
	 //My formula:  c_puct* value +  parent_count/vis_count   ~=	a*b*c/d
	thrust::transform(	
		thrust::make_zip_iterator(thrust::make_tuple(value_device.begin(), parent_visit_count_device.begin(), visit_count_device.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(value_device.end(), parent_visit_count_device.end(), visit_count_device.end())),
		result.begin(),
		DotProduct());

	//// method2
	////My formula : c_puct * value + parent_count / vis_count   ~= a * b * c / d

	//thrust::transform(
	//	parent_visit_count_device.begin(), parent_visit_count_device.end(),
	//	visit_count_device.begin(),
	//	temp_result.begin(), thrust::divides<float>());


	////c_puct* value + temp_result   ~= AX + B


	//	thrust::transform(
	//		value_device.begin(), value_device.end(),
	//		visit_count_device.begin(),
	//		visit_count_device.begin(),
	//		saxpy_functor(c_puct));



	//get_max_index
	thrust::device_vector<int>index(num);
	thrust::sequence(index.begin(), index.end());
	thrust::sort_by_key(result.begin(), result.end(), index.begin(), thrust::greater<float>());
	max_index = index[0];
	

}

void CUDA_accelerator::collect_data()
{
	thrust::host_vector<float>	value_host;
	thrust::host_vector<float>	visit_count_host;
	thrust::host_vector<float>	parent_visit_count_host;
	for (size_t i = 0; i < num; i++)
	{
		value_host.push_back((*working_on)[i]->value);
		visit_count_host.push_back((*working_on)[i]->visit_count);

		//ע�����޸��ڵ�
		parent_visit_count_host.push_back((*working_on)[i]->parent_node->visit_count);
	}
	value_device = value_host;
	visit_count_device = visit_count_host;
	parent_visit_count_device = parent_visit_count_host;
	result = thrust::device_vector<float>(num);
}
